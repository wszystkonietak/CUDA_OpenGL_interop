#include "hip/hip_runtime.h"
#include "Water.cuh"

struct FlipFluidConstants
{
	float2 resolution;
};

__global__ void create_solid_cells(hipSurfaceObject_t SurfObj, int width, int height)
{
	// Calculate surface coordinates
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	float2 data;
	if (x < width && y < height) {
		data.x = (x < 1 || x > width - 2) ? 0 : 1;
		data.y = (y < 1 || y > height - 2) ? 0 : 1;
		// Write to output surface
		surf2Dwrite(data, SurfObj, x * sizeof(float2), y);
	}
}

__global__ void clear_grid(hipSurfaceObject_t grid, uint2 resolution)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < resolution.x && y < resolution.y) {
		surf2Dwrite(0.0f, grid, x * sizeof(float), y);
	}
}

__global__ void simulate_particles(hipSurfaceObject_t grid, Particle* particles, 
	float2* grid_velocities, float* sum_of_weights, ushort2* busy_cells, unsigned int* busy_cells_size, 
	unsigned int particles_size, uint2 resolution, unsigned int grid_size, float4 boundings, float cell_size, 
	float delta_time) 
{
	extern __shared__ float2 s[];
	float2* s_grid_velocities = s;
	float* s_sum_of_weights = (float*)&s_grid_velocities[grid_size];
	unsigned short* s_busy_cells = (unsigned short*)&s_sum_of_weights[grid_size];
	
	int g_id = blockIdx.x * blockDim.x + threadIdx.x;
	int t_id = threadIdx.x;
	float2 vel, pos, delta;
	uint2 cell = make_uint2(0, 0), g_cell;
	float4 weights, prev_weights;

	for (int i = t_id; i < resolution.x * resolution.y; i += blockDim.x) {
		s_grid_velocities[i].x = 0;
		s_grid_velocities[i].y = 0;
		s_sum_of_weights[i] = 0;
		s_busy_cells[i] = 0;
	}
	__syncthreads();
	while (g_id < particles_size) {
		vel = particles[g_id].velocity;
		pos = particles[g_id].position;
		vel.y -= 3.81f * delta_time;
		
		pos.x += vel.x * delta_time;
		pos.y += vel.y * delta_time;

		if (pos.x <= boundings.x) {
			vel.x *= -0.8f;
			pos.x = boundings.x;
		}
		if (pos.x >= boundings.y) {
			vel.x *= -0.8f;
			pos.x = boundings.y;
		}

		if (pos.y <= boundings.z) {
			vel.y *= -0.8f;
			pos.y = boundings.z;
		}
		if (pos.y >= boundings.w) {
			vel.y *= -0.8f;
			pos.y = boundings.w;
		}

		particles[g_id].position = pos;
		cell = make_uint2(pos.x / cell_size, pos.y / cell_size);

		s_busy_cells[cell.x * resolution.y + cell.y]++;

		delta.x = (pos.x - cell_size * cell.x) / cell_size;
		delta.y = (pos.y - cell_size * cell.y) / cell_size;

		weights.x = (1 - delta.x) * (1 - delta.y);
		weights.y = delta.x * (1 - delta.y);
		weights.z = (delta.x) * (delta.y);
		weights.w = (1 - delta.x) * (delta.y);
		
		atomicAdd(&s_grid_velocities[cell.x * resolution.y + cell.y].x, weights.x * vel.x);
		atomicAdd(&s_grid_velocities[cell.x * resolution.y + cell.y].y, weights.x * vel.y);

		atomicAdd(&s_grid_velocities[(cell.x + 1) * resolution.y + cell.y].x, weights.y * vel.x);
		atomicAdd(&s_grid_velocities[(cell.x + 1) * resolution.y + cell.y].y, weights.y * vel.y);

		atomicAdd(&s_grid_velocities[(cell.x + 1) * resolution.y + cell.y + 1].x, weights.z * vel.x);
		atomicAdd(&s_grid_velocities[(cell.x + 1) * resolution.y + cell.y + 1].y, weights.z * vel.y);

		atomicAdd(&s_grid_velocities[cell.x * resolution.y + cell.y + 1].x, weights.w * vel.x);
		atomicAdd(&s_grid_velocities[cell.x * resolution.y + cell.y + 1].y, weights.w * vel.y);

		atomicAdd(&s_sum_of_weights[cell.x * resolution.y + cell.y], weights.x);
		atomicAdd(&s_sum_of_weights[(cell.x + 1) * resolution.y + cell.y], weights.y);
		atomicAdd(&s_sum_of_weights[(cell.x + 1) * resolution.y + cell.y + 1], weights.z);
		atomicAdd(&s_sum_of_weights[cell.x * resolution.y + cell.y + 1], weights.w);		

		g_id += blockDim.x * gridDim.x;
	}
	__syncthreads();

	g_cell.x = t_id / resolution.x;
	g_cell.y = t_id % resolution.y;

	while(g_cell.x < resolution.x)
	{
		atomicAdd(&sum_of_weights[t_id], s_sum_of_weights[t_id]);
		atomicAdd(&grid_velocities[t_id].x, s_grid_velocities[t_id].x);
		atomicAdd(&grid_velocities[t_id].y, s_grid_velocities[t_id].y);
		if (s_busy_cells[t_id])
			surf2Dwrite(1.0f, grid, g_cell.x * sizeof(float), g_cell.y);

		t_id += blockDim.x;
		g_cell.x = t_id / resolution.x;
		g_cell.y = t_id % resolution.y;
	}
}

__global__ void update_velocities(float2* grid_velocities, float* sum_of_weights, ushort2* busy_cells, unsigned int* busy_cells_size, uint2 resolution)
{
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
	float sum;
	float2 vel;
	if (x < resolution.x && y < resolution.y) {
		sum = sum_of_weights[x * resolution.y + y];
		if (sum > 0.0) {
			vel = grid_velocities[x * resolution.y + y];
			vel.x /= sum;
			vel.y /= sum;
			grid_velocities[x * resolution.y + y] = vel;
		}
	}
}

__global__ void calculate_divergence(hipSurfaceObject_t grid, hipSurfaceObject_t solid_cells, float2* grid_velocities, uint2 resolution, unsigned int iteration) {
	unsigned int x = blockIdx.x * blockDim.x + threadIdx.x + 1;
	unsigned int y = blockIdx.y * blockDim.y + threadIdx.y + 1;

	if (x < resolution.x - 1 && y < resolution.y - 1) {
		float isBusy = surf2Dread<float>(grid, x * sizeof(float), y);
		if (isBusy == 1.0f) {
			if ((x + y) % 2 == (iteration % 2)) {
				float left = grid_velocities[((x) * resolution.y) + y].x;
				float right = grid_velocities[((x + 1) * resolution.y) + y].x;
				float bottom = grid_velocities[(x * resolution.y) + (y)].y;
				float top = grid_velocities[(x * resolution.y) + (y + 1)].y;
				
				float solid_left = surf2Dread<float2>(solid_cells, (x - 1) * sizeof(float2), y).x;
				float solid_right = surf2Dread<float2>(solid_cells, (x + 1) * sizeof(float2), y).x;
				float solid_bottom = surf2Dread<float2>(solid_cells, x * sizeof(float2), y - 1).y;
				float solid_top = surf2Dread<float2>(solid_cells, x * sizeof(float2), y + 1).y;

				float sumOfStates = solid_left + solid_right + solid_bottom + solid_top;
				float divergence = (right - left + top - bottom);
				
				divergence *= 1.9f;
				divergence /= sumOfStates;
				grid_velocities[((x + 1) * resolution.y) + y].x -= divergence * solid_right;
				grid_velocities[((x) * resolution.y) + y].x += divergence * solid_left;
				grid_velocities[(x * resolution.y) + (y + 1)].y -= divergence * solid_top;
				grid_velocities[(x * resolution.y) + (y)].y += divergence * solid_bottom;
			}
		}
	}
}

__global__ void grid_to_particles(float2* grid_velocities, Particle* particles, float cell_size, int num_particles, uint2 resolution)
{
	int g_id = blockIdx.x * blockDim.x + threadIdx.x;
	float2 vel, pos, delta, vel_current, vel_right_bottom, vel_right_top, vel_left_top;
	uint2 cell;
	float4 weights;

	while (g_id < num_particles) {
		pos = particles[g_id].position;
		cell = make_uint2(pos.x / cell_size, pos.y / cell_size);


		vel_current = grid_velocities[cell.x * resolution.y + cell.y];
		vel_right_bottom = grid_velocities[(cell.x + 1) * resolution.y + cell.y];
		vel_right_top = grid_velocities[(cell.x + 1) * resolution.y + cell.y + 1];
		vel_left_top = grid_velocities[cell.x * resolution.y + cell.y + 1];

		delta.x = (pos.x - cell_size * cell.x) / cell_size;
		delta.y = (pos.y - cell_size * cell.y) / cell_size;

		weights.x = (1 - delta.x) * (1 - delta.y);
		weights.y = delta.x * (1 - delta.y);
		weights.z = (delta.x) * (delta.y);
		weights.w = (1 - delta.x) * (delta.y);

		vel.x = weights.x * vel_current.x;
		vel.x += weights.y * vel_right_bottom.x;
		vel.x += weights.z * vel_right_top.x;
		vel.x += weights.w * vel_left_top.x;

		vel.y = weights.x * vel_current.y;
		vel.y += weights.y * vel_right_bottom.y;
		vel.y += weights.z * vel_right_top.y;
		vel.y += weights.w * vel_left_top.y;

		particles[g_id].velocity = vel;

		g_id += blockDim.x * gridDim.x;
	}
}

void FlipFluid::init(std::string&& shaders_path)
{
	cell_size = 0.02;
	particle_radius = 0.0033;
	s_textures = Shader(shaders_path + "/canvas.vert", shaders_path + "/canvas.frag");
	s_particles = Shader(shaders_path + "/particles.vert", shaders_path + "/particles.frag");
	s_particles.use();
	s_particles.setMat4("u_projectionViewMatrix", glm::ortho(0.0f, 1.0f, 0.0f, 1.0f, -1.0f, 1.0f));
	s_particles.setFloat("u_inPixelDiameter", particle_radius * 1000);
	if (boundings.x > boundings.y) {
		std::swap(boundings.x, boundings.y);
	}
	if (boundings.z > boundings.w) {
		std::swap(boundings.z, boundings.w);
	}
	size.x = boundings.y - boundings.x;
	size.y = boundings.w - boundings.z;
	resolution = make_uint2(size.x / cell_size, size.y / cell_size);
	particles_size = size.x * size.y * rest_particle_density;
	particles_size = 10000;
	particle_boundings = make_float4(boundings.x + cell_size + 0.00001, boundings.y - cell_size - 0.00001, boundings.z + cell_size + 0.00001, boundings.w - cell_size - 0.00001);
	mem_size = resolution.x * resolution.y;

	glGenTextures(1, &id_solid_cells);
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, id_solid_cells);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_RG32F, resolution.x, resolution.y, 0, GL_RG, GL_FLOAT, NULL);
	glBindImageTexture(0, id_solid_cells, 0, GL_FALSE, 0, GL_READ_WRITE, GL_RG32F);

	glGenTextures(1, &id_grid);
	glActiveTexture(GL_TEXTURE2);
	glBindTexture(GL_TEXTURE_2D, id_grid);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
	glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
	glTexImage2D(GL_TEXTURE_2D, 0, GL_R32F, resolution.x, resolution.y, 0, GL_RED, GL_FLOAT, NULL);
	glBindImageTexture(0, id_grid, 0, GL_FALSE, 0, GL_READ_WRITE, GL_R32F);
	
	hipMalloc(&d_busy_cells, sizeof(ushort2) * mem_size);
	hipMalloc(&d_busy_cells_size, sizeof(unsigned int));

	hipMalloc(&d_grid_velocities, sizeof(float2) * mem_size);
	hipMalloc(&d_sum_of_weights, sizeof(float) * mem_size);
	

	solid_cells = Surface<float2>(id_solid_cells, GL_TEXTURE_2D, resolution);
	grid = Surface<float>(id_grid, GL_TEXTURE_2D, resolution);

	
	block_size = dim3(32, 32);
	grid_size = dim3(((unsigned int)resolution.x + block_size.x - 1) / block_size.x, 
		((unsigned int)resolution.y + block_size.y - 1) / block_size.y);
	p_block_size = dim3(1024);
	p_grid_size = dim3((particles_size - 1) / p_block_size.x + 1);
	create_solid_cells<< <grid_size, block_size >> > 
		(solid_cells.surface, (unsigned int)resolution.x, (unsigned int)resolution.y);
	particles = std::vector<Particle>(particles_size);


	std::uniform_real_distribution<float> rand_width(boundings.x + cell_size, 
		(boundings.y - cell_size) / 2.0);
	std::uniform_real_distribution<float> rand_height(boundings.z + cell_size, 
		boundings.w - cell_size);
	std::uniform_real_distribution<float> rand_velocity(-1, 1);
	for (int i = 0; i < particles_size; i++) {
		particles[i].position = make_float2(rand_width(gen), rand_height(gen));
		particles[i].velocity = make_float2(rand_velocity(gen), rand_velocity(gen));
	}

	glGenVertexArrays(1, &VAO);
	glGenBuffers(1, &VBO);
	glBindVertexArray(VAO);

	glBindBuffer(GL_ARRAY_BUFFER, VBO);
	glBufferData(GL_ARRAY_BUFFER, particles.size() * sizeof(Particle), &particles[0], GL_DYNAMIC_DRAW);

	glEnableVertexAttribArray(0);
	glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, sizeof(Particle), (void*)offsetof(Particle, position));

	glEnableVertexAttribArray(1);
	glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, sizeof(Particle), (void*)offsetof(Particle, velocity));

	glBindBuffer(GL_ARRAY_BUFFER, 0);

	glBindVertexArray(0);

	size_t num_bytes;
	hipGraphicsResource* cuda_vbo_resource;
	hipGraphicsGLRegisterBuffer(&cuda_vbo_resource, VBO, cudaGraphicsMapFlagsNone);
	hipGraphicsMapResources(1, &cuda_vbo_resource, 0);
	hipGraphicsResourceGetMappedPointer((void**)&d_particles, &num_bytes, cuda_vbo_resource);

	hipGraphicsUnmapResources(1, &cuda_vbo_resource);
	collisions.setup(size.x, size.y, particles.size(), particle_radius, d_particles);
	hipDeviceSynchronize();
	simulate_particles_shared_size = sizeof(float2) * mem_size + 
		sizeof(float) * mem_size + sizeof(unsigned short) * mem_size;
}

void FlipFluid::update()
{
	for (int i = 0; i < 5; i++) {
		collisions.check_collision();
	}
	hipMemset( d_busy_cells, 0, sizeof(ushort2) * mem_size);
	hipMemset(d_busy_cells_size, 0, sizeof(unsigned int));

	hipMemset(d_grid_velocities, 0, sizeof(float2) * mem_size);
	hipMemset(d_sum_of_weights, 0, sizeof(float) * mem_size);
	
	clear_grid << <grid_size, block_size >> > (grid.surface, resolution);
	
	simulate_particles <<< p_grid_size, p_block_size, simulate_particles_shared_size >>> (grid.surface, d_particles, d_grid_velocities, d_sum_of_weights, d_busy_cells, d_busy_cells_size, particles_size, resolution, mem_size, particle_boundings, cell_size, Time::delta_time);

	update_velocities <<<grid_size, block_size >>> (d_grid_velocities, d_sum_of_weights, d_busy_cells, d_busy_cells_size, resolution);
	
	for (int i = 0; i < 40; i++) {
		calculate_divergence << <grid_size, block_size >> > (grid.surface, solid_cells.surface, d_grid_velocities, resolution, i);
	}

	grid_to_particles << <p_grid_size, p_block_size >> > (d_grid_velocities, d_particles, cell_size, particles_size, resolution);
	hipDeviceSynchronize();
}

void FlipFluid::draw()
{
	hipDeviceSynchronize();
	s_textures.use();
	
	unsigned int quadvao = 0, quadvbo = 0;
	if (quadvao == 0) {
		float quadVertices[] = {
			// positions  // texture Coords
			-1.0f,  1.0f, 0.0f, 1.0f,
			-1.0f, -1.0f, 0.0f, 0.0f,
			 1.0f,  1.0f, 1.0f, 1.0f,
			 1.0f, -1.0f, 1.0f, 0.0f,
		};
		// setup plane VAO
		glGenVertexArrays(1, &quadvao);
		glGenBuffers(1, &quadvbo);
		glBindVertexArray(quadvao);
		glBindBuffer(GL_ARRAY_BUFFER, quadvbo);
		glBufferData(GL_ARRAY_BUFFER, sizeof(quadVertices), &quadVertices, GL_STATIC_DRAW);
		glEnableVertexAttribArray(0);
		glVertexAttribPointer(0, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)0);
		glEnableVertexAttribArray(1);
		glVertexAttribPointer(1, 2, GL_FLOAT, GL_FALSE, 4 * sizeof(float), (void*)(2 * sizeof(float)));
	}
	
	glActiveTexture(GL_TEXTURE0);
	glBindTexture(GL_TEXTURE_2D, id_grid);
	glBindVertexArray(quadvao);
	glDrawArrays(GL_TRIANGLE_STRIP, 0, 4);

	s_particles.use();    
	glBindVertexArray(VAO);
	glDrawArrays(GL_POINTS, 0, particles.size());
	glBindVertexArray(0);
}
