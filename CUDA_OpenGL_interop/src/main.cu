#include <hip/hip_runtime.h>
#include <stdio.h>

#include <glad/glad.h>
#include <GLFW/glfw3.h>

#include <iostream>

int main()
{
    int deviceCount;
    hipError_t error = hipGetDeviceCount(&deviceCount);

    if (error != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(error));
        return 1;
    }

    printf("CUDA device count: %d\n", deviceCount);
    
    glfwInit();
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    GLFWwindow* window = glfwCreateWindow(800, 800, "GLFWwindow", NULL, NULL);
    if (window == NULL)
    {
        glfwTerminate();
        return -1;
    }
    glfwMakeContextCurrent(window);

    if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress))
    {
        std::cout << "Failed to initialize GLAD" << std::endl;
        return -1;
    }
    while (!glfwWindowShouldClose(window))
    {
        glClearColor(0.3f, 0.4f, 0.5f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT);
        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    glfwTerminate();
    return 0;
}